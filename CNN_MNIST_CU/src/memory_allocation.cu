#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include "memory_allocation.h"

void AllocateAndCopyMemory(double* dev_flattened_kernels, const double* flattened_kernels, size_t size) {
    
    hipMalloc((void**)&dev_flattened_kernels, size * sizeof(double));
    hipMemcpy(dev_flattened_kernels, flattened_kernels, size * sizeof(double), hipMemcpyHostToDevice);

    // Check for errors
    hipDeviceSynchronize();
    if (hipGetLastError() != hipSuccess) {
        std::cout << "Error copying memory to GPU: " << hipGetErrorString(hipGetLastError()) << std::endl;
    } else {
        std::cout << "Memory successfully copied to GPU." << std::endl;
    }
}

void FreeMemory(double* dev_ptr) {
    hipError_t cudaStatus = hipFree(dev_ptr);
    
    if (cudaStatus != hipSuccess) {
        std::cout << "Error freeing GPU memory: " << hipGetErrorString(cudaStatus) << std::endl;
    } else {
        std::cout << "GPU memory successfully freed." << std::endl;
    }
}
