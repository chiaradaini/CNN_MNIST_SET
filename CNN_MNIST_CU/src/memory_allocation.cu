#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include "memory_allocation.h"

void AllocateMemory(double** dev_flattened_kernels, const double* flattened_kernels, size_t size) {
    hipMalloc((void**)dev_flattened_kernels, size * sizeof(double));

    // Check for errors
    hipDeviceSynchronize();
    if (hipGetLastError() != hipSuccess) {
        std::cout << "Error allocating memory on the GPU: " << hipGetErrorString(hipGetLastError()) << std::endl;
    } else {
        std::cout << "Memory successfully allocated on GPU." << std::endl;
    }
}

void CopyMemoryToDevice(double** dev_flattened_kernels, const double* flattened_kernels, size_t size) {
    hipMemcpy(*dev_flattened_kernels, flattened_kernels, size * sizeof(double), hipMemcpyHostToDevice);

    // Check for errors
    hipDeviceSynchronize();
    if (hipGetLastError() != hipSuccess) {
        std::cout << "Error copying data to GPU: " << hipGetErrorString(hipGetLastError()) << std::endl;
    } else {
        std::cout << "Data successfully copied to GPU." << std::endl;
    }
}

void CopyMemoryToHost(double* host_data, double** dev_data, size_t size) {
    hipMemcpy(host_data, *dev_data, size * sizeof(double), hipMemcpyDeviceToHost);

    // Check for errors
    hipDeviceSynchronize();
    if (hipGetLastError() != hipSuccess) {
        std::cout << "Error copying data from GPU: " << hipGetErrorString(hipGetLastError()) << std::endl;
    } else {
        std::cout << "Data successfully copied from GPU to host." << std::endl;
    }
}

void FreeMemory(double* dev_ptr) {
    hipError_t cudaStatus = hipFree(dev_ptr);

    // Check for errors
    // if (cudaStatus != hipSuccess) {
    //     std::cout << "Error freeing GPU memory: " << hipGetErrorString(cudaStatus) << std::endl;
    // } else {
    //     std::cout << "GPU memory successfully freed." << std::endl;
    // }
}
