#include "hip/hip_runtime.h"
#include <vector>
#include <limits>
#include <vector>
#include <limits>
#include <cstdlib>
#include <tuple>
#include <random>
#include "Layer.h"
#include "Functions.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include "cuda_error_check.h"

__global__ void convolution(const double* input, const double* kernels, int kernel_size, int input_h, int input_w, int output_h, int output_w, int input_channels, int output_channels, double* conv_output) {
	    //printf("block Idx = %d, block Dim = %d, thread Idx = %d\n", blockIdx.x, blockDim.x, threadIdx.x);
	    //printf("block Idx = %d, block Dim = %d, thread Idx = %d\n", blockIdx.y, blockDim.y, threadIdx.y);
            //printf("block Idx = %d, block Dim = %d, thread Idx = %d\n", blockIdx.z, blockDim.z, threadIdx.z);
	    int c = blockIdx.z * blockDim.z + threadIdx.z;
	    int h = blockIdx.y * blockDim.y + threadIdx.y;
	    int w = blockIdx.x * blockDim.x + threadIdx.x;
	    int output_idx = (c * output_h + h) * output_w + w;

//	    clock_t start_add = clock64();
//	    clock_t end_add = clock64();
	    if (h < output_h && w < output_w) {
		double sum = 0.0;
		for (int i = 0; i < kernel_size; ++i) {
			for (int j = 0; j < kernel_size; ++j) {
				for (int k = 0; k < input_channels; ++k) {
                			double input_pixel = input[(h + i) * (input_w * input_channels) + (w + j) * input_channels + k];
			                double kernel_value = kernels[c * (kernel_size * kernel_size * input_channels) + i * (kernel_size * input_channels) + j * input_channels + k];
					sum += input_pixel * kernel_value;
					//printf("c = %d, h = %d, w = %d, output_idx = %d, input = %f, kernel = %f, sum = %f\n", c, h, w, output_idx, input_pixel, kernel_value, sum);
					}
				}
			}
//		unsigned long micro_add = (end_add - start_add) * 1000000 / CLOCKS_PER_SEC;
//		printf("start = %llu, end = %llu, elapsed time = %llu [micro s]\n", start_add, end_add, micro_add);
		conv_output[output_idx] = sum;
		}
}

image3D conv_forward_prop(const image3D& input, const image3D& kernels, const int& numBlocks) {
    // Allocate memory on the GPU
    double *dev_conv_output;
    int input_h = input.size();
    int input_w = input[0].size();
    int input_channels = input[0][0].size();
    int kernel_size = kernels[0].size();
    int output_channels = kernels.size();
    int output_h = input_h - kernel_size + 1;
    int output_w = input_w - kernel_size + 1;
    //size_t input_size = input_h * input_w * input_channels * sizeof(double);
    //size_t kernels_size = output_channels * kernel_size * kernel_size * input_channels * sizeof(double);
    size_t conv_output_size = output_h * output_w * output_channels * sizeof(double);
    
    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    size_t usedMem = totalMem - freeMem;
    //printf("Used GPU memory before allocation: %zu bytes\n", usedMem);

    //hipMalloc((void**)&dev_input, input_size);
    //hipMalloc((void**)&dev_kernels, kernels_size);
    hipMalloc((void**)&dev_conv_output, conv_output_size);
    
    hipMemGetInfo(&freeMem, &totalMem);
    usedMem = totalMem - freeMem;
    //printf("Used GPU memory after allocation: %zu bytes\n", usedMem);

    image1D flattened_input = convert_to_flattened_input(input);
    image1D flattened_kernels = convert_to_flattened_input(kernels);

    double *dev_flattened_input, *dev_flattened_kernels;
    hipMalloc((void**)&dev_flattened_input, flattened_input.size() * sizeof(double));
    hipMalloc((void**)&dev_flattened_kernels, flattened_kernels.size() * sizeof(double));

    hipMemcpy(dev_flattened_input, flattened_input.data(), flattened_input.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_flattened_kernels, flattened_kernels.data(), flattened_kernels.size() * sizeof(double), hipMemcpyHostToDevice);

    dim3 blockDim(numBlocks, numBlocks);
    dim3 gridDim((output_w + blockDim.x - 1) / blockDim.x, (output_h + blockDim.y - 1) / blockDim.y, output_channels);

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start));

    convolution<<<blockDim, gridDim>>>(dev_flattened_input, dev_flattened_kernels, kernel_size,
                                                input_h, input_w, output_h, output_w, input_channels, output_channels, dev_conv_output);


    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Kernel execution time: %.2f ms\n", milliseconds);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipDeviceSynchronize();

    image1D conv_output_host(output_h * output_w * output_channels);
    hipMemcpy(conv_output_host.data(), dev_conv_output, conv_output_size, hipMemcpyDeviceToHost);

    hipFree(dev_flattened_input);
    hipFree(dev_flattened_kernels);
    hipFree(dev_conv_output);

    hipMemGetInfo(&freeMem, &totalMem);
    usedMem = totalMem - freeMem;
    //printf("Used GPU memory after deallocation: %zu bytes\n", usedMem);
    
    image3D conv_output(output_h, image2D(output_w, image1D(output_channels)));
    conv_output = convertTo3D(conv_output_host, output_h, output_w, output_channels);

    //print_kernels(conv_output);
    return conv_output;

}

